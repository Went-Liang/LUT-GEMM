#include "hip/hip_runtime.h"
#include <omp.h>
#include <iostream>
#include <Eigen/Core>
#include <ctime>
#include <cstdlib>
#include <unsupported/Eigen/CXX11/Tensor>


template<typename T, typename STYPE, typename BTYPE>
void data_generator(const int q, const int g,
                    const unsigned m, const unsigned n, const unsigned k,
                    Eigen::Tensor<T, 2, Eigen::RowMajor>& W, Eigen::Tensor<T, 2, Eigen::RowMajor>& X, Eigen::Tensor<T, 2, Eigen::RowMajor>& Y,
                    Eigen::Tensor<STYPE, 3, Eigen::RowMajor>& A, Eigen::Tensor<BTYPE, 3, Eigen::RowMajor>& B){
    srand((unsigned)time(NULL));
    int subgroup_size = sizeof(BTYPE) * 8;
    Eigen::Tensor<STYPE, 3, Eigen::RowMajor> bitB(m, n, q);
    Eigen::Tensor<STYPE, 3, Eigen::RowMajor> bitB_(m, n, q);
    Eigen::Tensor<STYPE, 3, Eigen::RowMajor> completeA(m, n, q);

    X.setRandom();
    A.setRandom();
//    X.setConstant(1);
//    A.setConstant(1);

    for(unsigned i = 0; i < m; ++i){
        for(unsigned j = 0; j < n; ++j){
            Eigen::array<Eigen::Index, 3> offsets = {i, j, 0};
            Eigen::array<Eigen::Index, 3> offsets_ = {i, (int)ceil(j / g), 0};
            Eigen::array<Eigen::Index, 3> extents = {1, 1, q};
            completeA.slice(offsets, extents) = A.slice(offsets_, extents);
        }
    }
    for(unsigned i = 0; i < m; ++i){
        for(unsigned j = 0; j < n; ++j){
            for(unsigned z = 0; z < q; ++z) {
                bitB(i, j, z) = (STYPE)((rand()%2) > 0.5? 1.0: -1.0);
                bitB_(i, j, z) = (STYPE)(bitB(i, j, z) == 1? 1.0: 0.0);
            }
        }
    }

    Eigen::Tensor<STYPE, 1> powlist(subgroup_size);
    for(int i = 0; i < subgroup_size; ++i)
        powlist(i) = (STYPE)pow(2, i);

    Eigen::array<Eigen::IndexPair<int>, 1> product_dims = { Eigen::IndexPair<int>(0, 0) };
    for(unsigned i = 0; i < m; ++i){
        for(unsigned j = 0; j < (int)ceil((float)n / (float)subgroup_size); ++j){
            for(unsigned z = 0; z < q; ++z) {
                BTYPE sum = 0;
                for(unsigned c = 0; c < subgroup_size; ++c) {
                    if(j * subgroup_size + c >= n)continue;
                    sum += (BTYPE) (powlist(c) * bitB_(i, j * subgroup_size + c, z));
                }
                B(i, j, z) = sum;
            }
        }
    }

    auto Z = completeA * bitB;
    Eigen::array<int, 1> dims({2});
    W = Z.sum(dims);


    product_dims = Eigen::IndexPair<int>(1, 0);
    Y = W.contract(X, product_dims);

//    std::cout << bitB << "\n";
}


template<typename T, typename STYPE, typename BTYPE>
void checkAB(const int q, const int g,
             const unsigned m, const unsigned n, const unsigned k,
             const Eigen::Tensor<T, 2, Eigen::RowMajor> X, const Eigen::Tensor<T, 2, Eigen::RowMajor> Y,
             const Eigen::Tensor<STYPE, 3, Eigen::RowMajor> A, const Eigen::Tensor<BTYPE, 3, Eigen::RowMajor> B){
    int subgroup_size = sizeof(BTYPE) * 8;
    Eigen::Tensor<T, 2, Eigen::RowMajor> res(m, k);
    res.setZero();

    for(int iq = 0; iq < q; ++iq){
        for(int i = 0; i < m; ++i){
            for(int j = 0; j < n; ++j){
                for(int ik = 0; ik < k; ++ik) {
                    int group_id = (int) ceil(j / g);
                    STYPE a = A(i, group_id, iq);

                    int subgroup_id = j / subgroup_size;
                    BTYPE decimal_b = B(i, subgroup_id, iq);
                    int bin_id = j % subgroup_size;
                    BTYPE shifted = decimal_b >> bin_id;
                    int bitValue = shifted & 1;
                    STYPE b = (STYPE)(bitValue == 1 ? 1 : -1);

                    T abx = ((T)(a * b)) * X(j, ik);
                    res(i, ik) += abx;
                }
            }
        }
    }
    Eigen::Tensor<T, 2, Eigen::RowMajor> diffArray = (res - Y).abs();
    std::cout << "Max Error: " << diffArray.maximum() << "\n";
}

template<typename T, typename STYPE, typename BTYPE>
void checkAB_(const int q, const int g,
             const unsigned m, const unsigned n, const unsigned k,
              const Eigen::Tensor<T, 2, Eigen::RowMajor> X, const Eigen::Tensor<T, 2, Eigen::RowMajor> Y,
              const Eigen::Tensor<STYPE, 3, Eigen::RowMajor> A, const Eigen::Tensor<BTYPE, 3, Eigen::RowMajor> B){
    int subgroup_size = sizeof(BTYPE) * 8;
    Eigen::Tensor<T, 2, Eigen::RowMajor> res(m, k);
    res.setZero();
    // B -> bitB
    // (A * B) * X

    for(int iq = 0; iq < q; ++iq){
        for(int i = 0; i < m; ++i){
            for(int j = 0; j < k; ++j){
                for(int in = 0; in < n; ++in) {
                    int group_id = (int) ceil(in / g);
                    STYPE a = A(i, group_id, iq);

                    int subgroup_id = in / subgroup_size;
                    BTYPE decimal_b = B(i, subgroup_id, iq);
                    int bin_id = in % subgroup_size;
                    BTYPE shifted = decimal_b >> bin_id;
                    int bitValue = shifted & 1;
                    STYPE b = (STYPE)(bitValue == 1 ? 1 : -1);

                    T abx = ((T)(a * b)) * X(in, j);
                    res(i, j) += abx;
                }
            }
        }
    }
    Eigen::Tensor<T, 2, Eigen::RowMajor> diffArray = (res - Y).abs();
    std::cout << "Max Error: " << diffArray.maximum() << "\n";
}

template<typename T, typename STYPE, typename BTYPE>
void checkAB_1(const int q, const int g,
               const unsigned m, const unsigned n, const unsigned k,
               const Eigen::Tensor<T, 2, Eigen::RowMajor> X, const Eigen::Tensor<T, 2, Eigen::RowMajor> Y,
               const Eigen::Tensor<STYPE, 3, Eigen::RowMajor> A, const Eigen::Tensor<BTYPE, 3, Eigen::RowMajor> B){
    int subgroup_size = sizeof(BTYPE) * 8;
    Eigen::Tensor<T, 2, Eigen::RowMajor> res(m, k);
    res.setZero();
    // B -> bitB
    // (A * B) * X
    for(int i = 0; i < m; ++i){
        for(int j = 0; j < k; ++j){

            for(int in = 0; in < n; ++in) {
                T w{0};
                for(int iq = 0; iq < q; ++iq){
                    int group_id = (int) ceil(in / g);
                    STYPE a = A(i, group_id, iq);

                    int subgroup_id = in / subgroup_size;
                    BTYPE decimal_b = B(i, subgroup_id, iq);
                    int bin_id = in % subgroup_size;
                    BTYPE shifted = decimal_b >> bin_id;
                    int bitValue = shifted & 1;
                    STYPE b = (STYPE)(bitValue == 1 ? 1 : -1);
                    w += a * b;
                }

                T abx = w * X(in, j);
                res(i, j) += abx;
            }
        }
    }
    Eigen::Tensor<T, 2, Eigen::RowMajor> diffArray = (res - Y).abs();
    std::cout << "Max Error: " << diffArray.maximum() << "\n";
}


template<typename T, typename STYPE, typename BTYPE>
__global__ void lut_gemm_kernel(const int q, const int group_size,
                                const unsigned m, const unsigned n, const unsigned k,
                                const unsigned A_col_num, const unsigned B_col_num,
                                const STYPE *A, const BTYPE *B, const T *X,
                                T *Y) {
// grid(m/th, n/g), block(th)
// W(m*n), X(n*k), Y(m*k), AB(m*n*q)
// m = 2, n = 8, g = 8,
    constexpr unsigned int subgroup_size = sizeof(BTYPE) * 8;
    unsigned int lut_num = group_size / subgroup_size;
    unsigned int lut_size = pow(2, subgroup_size);
    unsigned int group_id = blockIdx.y;
    unsigned int row_id = threadIdx.x + blockDim.x * blockIdx.x;
    extern __shared__ T luts[];

    for(int ik = 0; ik < k; ++ik) {
        if (threadIdx.x < lut_size) {
            for (int i = 0; i < lut_num; ++i) {
                T sum = 0;
#pragma unroll
                for (int j = 0; j < subgroup_size; ++j) {
                    sum += X[(group_id * group_size + i * subgroup_size + j) * k + ik] *
                           (((threadIdx.x % lut_size >> j) & 1) ? 1 : -1);
                }
                luts[i * lut_size + threadIdx.x % lut_size] = sum;
            }
        }
        __syncthreads();

        for (int iq = 0; iq < q; ++iq) {
            T sum = 0;
            for (int i = 0; i < lut_num; ++i) {
                BTYPE decimal_b = B[row_id * B_col_num * q + (group_id * lut_num + i) * q + iq];
                sum += luts[i * lut_size + decimal_b];
            }
            sum *= A[row_id * A_col_num * q + group_id * q + iq];
            atomicAdd(&Y[row_id * k + ik], sum);
        }
    }
}

int main() {
    int gpu_rank = 0;
    hipDeviceProp_t deviceProp{};
    hipGetDeviceProperties(&deviceProp, gpu_rank);
    hipSetDevice(gpu_rank);
    printf("GPU %s status: ", deviceProp.name);
    double boostFrequency = deviceProp.clockRate / 1e6;
    int fp32CoresNum = 640;
    double peakPerformance = boostFrequency * fp32CoresNum * 2;
    printf("clock rate %.3f GHz, FP32 cores num %d, FP32 peak throughput %.3f "
            "GFLOPS\n",
            boostFrequency, fp32CoresNum, peakPerformance);
    omp_set_num_threads(omp_get_num_procs());


//    typedef Eigen::bfloat16 T;        // W X Y: before quant
    typedef float T;                    // W X Y: before quant
    typedef float STYPE;                // scaleMat  A
    typedef std::uint8_t BTYPE;         // binaryMat B
    int subgroup_size = sizeof(BTYPE) * 8;
    int q = 3;
    int g = 128;
    unsigned m = 4096;
    unsigned n = 1024;
    unsigned k = 8;
    unsigned A_col_num = (int)ceil((float)n / (float)g);
    unsigned B_col_num = (int)ceil((float)n / (float)subgroup_size);

    Eigen::Tensor<T, 2, Eigen::RowMajor> W(m, n), X(n, k), Y(m, k);
    Eigen::Tensor<STYPE, 3, Eigen::RowMajor> A(m, A_col_num, q);
    Eigen::Tensor<BTYPE, 3, Eigen::RowMajor> B(m, B_col_num, q);

    data_generator<T, STYPE, BTYPE>(q, g, m, n, k,
                                    W, X, Y, A, B);
    checkAB_1<T, STYPE, BTYPE>(q, g, m, n, k,
                               X, Y, A, B);


    STYPE *deviceAPtr;
    BTYPE *deviceBPtr;
    T *deviceXPtr;
    T *deviceYPtr;
    hipMalloc(&deviceAPtr, (m * A_col_num * q) * sizeof(STYPE));
    hipMalloc(&deviceBPtr, (m * B_col_num * q) * sizeof(BTYPE));
    hipMalloc(&deviceXPtr, (n * k) * sizeof(T));
    hipMalloc(&deviceYPtr, (m * k) * sizeof(T));
    hipMemcpy(deviceAPtr, A.data(), (m * A_col_num * q) * sizeof(STYPE),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceBPtr, B.data(), (m * B_col_num * q) * sizeof(BTYPE),
               hipMemcpyHostToDevice);
    hipMemcpy(deviceXPtr, X.data(), (n * k) * sizeof(T),
               hipMemcpyHostToDevice);
    hipMemset(deviceYPtr, 0, (m * k) * sizeof(T));
    Eigen::Tensor<T, 2, Eigen::RowMajor> cuY(m, k);

    int th = 1024;
    dim3 block(th);
    dim3 grid((m - 1) / block.x + 1, (n - 1) / g + 1);

    lut_gemm_kernel<T, STYPE, BTYPE>
            <<<grid, block, pow(2, subgroup_size) * (g / subgroup_size) * sizeof(T)>>>
            (q, g, m, n, k,
             A_col_num, B_col_num,
             deviceAPtr, deviceBPtr, deviceXPtr, deviceYPtr);

    hipDeviceSynchronize();
    hipMemcpy(cuY.data(), deviceYPtr, (m * k) * sizeof(T), hipMemcpyDeviceToHost);
    Eigen::Tensor<T, 2, Eigen::RowMajor> diffArray = (cuY - Y).abs();
    std::cout << "lut-gemm Max Error: " << diffArray.maximum() << "\n";


//    std::cout << W << std::endl;
//    std::cout << "\n";
//    std::cout << X << std::endl;
//    std::cout << "\n";
//    std::cout << Y << std::endl;
//    std::cout << "\n";
//    std::cout << A << std::endl;
//    std::cout << "\n";
//    std::cout << B << std::endl;
//    std::cout << "\n";
//    std::cout << cuY << std::endl;


}
